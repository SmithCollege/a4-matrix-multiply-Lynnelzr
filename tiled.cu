#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matmul_tiled(float* A, float* B, float* C, int N) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            tile_A[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0;

        if (col < N && t * TILE_SIZE + threadIdx.y < N)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 512;
    size_t size = N * N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    // Timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_tiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Tiled Matrix Multiply Time: %.6f ms\n", milliseconds);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

